#include "hip/hip_runtime.h"
#include "Matrix.cuh"

struct Comparator {
    __host__ __device__ bool operator()(double num1, double num2) {
        return fabs(num1) < fabs(num2);
    }
};

__global__ void swapRows (double *data, uint64_t n, uint64_t i, uint64_t j) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetX = gridDim.x * blockDim.x;

    double tmp;
    for (uint64_t k = idx; k < 2 * n; k += offsetX) {
        tmp = data[k * n + i];
        data[k * n + i] = data[k * n + j];
        data[k * n + j] = tmp;
    }
}

__global__ void normalisation (double *data, uint64_t n, uint64_t i) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetX = gridDim.x * blockDim.x;

    //double coeff = data[n * i + i];
    for (uint64_t k = idx + i + 1; k < 2 * n; k += offsetX) {
        data[k * n + i] /= data[i * n + i];;
    }
}

__global__ void iteration (double *data, uint64_t n, uint64_t id) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetX = gridDim.x * blockDim.x;
    int offsetY = gridDim.y * blockDim.y;

    for (uint64_t i = idx + id + 1; i < 2 * n; i += offsetX) {
        //double coeff = data[id * n + i];
        for (uint64_t j = idy + id + 1; j < n; j += offsetY) {
            data[i * n + j] -= data[i * n + id] * data[id * n + j];
        }
    }
}

// __global__ void backIteration (double *data, uint64_t n, uint64_t id) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int idy = blockIdx.y * blockDim.y + threadIdx.y;
//     int offsetX = gridDim.x * blockDim.x;
//     int offsetY = gridDim.y * blockDim.y;

//     for (uint64_t i = idx + id; i < n; i += offsetX) {
//         double coeff = data[n * (n - id + 1) - i - 1] / data[n * (n - id + 1) - id];
//         for (uint64_t j = idy; j < 2 * n; j += offsetY) {
//             data[n * (j + 1) - i - 1] -= data[n * (j + 1) - id] * coeff;
//         }
//     }
// }

__global__ void backIteration (double *data, uint64_t n, uint64_t id) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offsetX = gridDim.x * blockDim.x;
    int offsetY = gridDim.y * blockDim.y;

    for (uint64_t i = idx + id + 1; i < 2 * n; i += offsetX) {
        //double coeff = data[n * (n - id + 1) - i - 1] / data[n * (n - id + 1) - id];
        //double coeff = 1;
        for (uint64_t j = idy; j <= id - 1; j += offsetY) {
            //data[n * (j + 1) - i - 1] -= data[n * (j + 1) - id] * coeff;
            data[i * n + j] -= data[i * n + id] * data[id * n + j];
        }
    }
}

Matrix::Matrix () : n(0), m(0) {}

Matrix::Matrix (uint64_t size) : n(size), m(size), data(n * m, 0) {
    for (uint64_t i = 0; i < size; ++i) {
        data[i * size + i] = 1;
    }
}

Matrix::Matrix (Matrix const &matrix) {
    n = matrix.n;
    m = matrix.m;
    data = matrix.data;
}

Matrix::Matrix (Matrix &&matrix) {
    n = matrix.n;
    m = matrix.m;
    data = std::move(matrix.data);
}

Matrix::Matrix (uint64_t n, uint64_t m, const std::vector<double> &buff) : n(n), m(m), data(buff) {}

Matrix::Matrix (uint64_t n, uint64_t m, double *buff) : n(n), m(m) {
    data.resize(n * m);
    data.assign(buff, buff + n * m);
}

Matrix::Matrix (uint64_t n, uint64_t m) : n(n), m(m), data(n * m, 0) {}

Matrix::~Matrix () {}

double &Matrix::operator() (uint64_t i, uint64_t j) {
    return data[i * m + j];
}

double Matrix::operator() (uint64_t i, uint64_t j) const {
    return data[i * m + j];
}

Matrix &Matrix::operator= (const Matrix &matrix) {
    n = matrix.n;
    m = matrix.m;
    data = matrix.data;
    return *this;
}

Matrix &Matrix::operator= (Matrix &&matrix) {
    n = matrix.n;
    m = matrix.m;
    data = std::move(matrix.data);
    return *this;
}

Matrix Matrix::reverse () const {
    Matrix ans(n), tmp(2 * n, n);
    double *data;

    for (uint64_t i = 0; i < n; ++i) {
        for (uint64_t j = 0; j < n; ++j) {
            tmp(j, i) = this->operator()(i, j);
        }
    }
    for (uint64_t i = 0; i < n; ++i) {
        tmp(i + n, i) = 1;
    }

    gpuErrorCheck(hipMalloc(&data, sizeof(double) * n * n * 2));
    gpuErrorCheck(hipMemcpy(data, tmp.data.data(), sizeof(double) * n * n * 2, hipMemcpyHostToDevice));

    Comparator check;

    for (uint64_t i = 0; i < n - 1; ++i) {
        thrust::device_ptr<double> device_data = thrust::device_pointer_cast(data + i * n);
        thrust::device_ptr<double> max = thrust::max_element(device_data + i, device_data + n, check);
        uint64_t idx = max - device_data;

        if (i != idx) {
            swapRows<<<256, 256>>>(data, n, i, idx);
            gpuErrorCheck(hipGetLastError());
            gpuErrorCheck(hipDeviceSynchronize());
        }

        normalisation<<<256, 256>>>(data, n, i);
        gpuErrorCheck(hipGetLastError());
        gpuErrorCheck(hipDeviceSynchronize());

        iteration<<<256, 256>>>(data, n, i);
        gpuErrorCheck(hipGetLastError());
        gpuErrorCheck(hipDeviceSynchronize());
    }
    normalisation<<<256, 256>>>(data, n, n - 1);
    gpuErrorCheck(hipGetLastError());
    gpuErrorCheck(hipDeviceSynchronize());

    for (uint64_t i = n - 1; i > 0; --i) {
        backIteration<<<256, 256>>>(data, n, i);
        gpuErrorCheck(hipGetLastError());
        gpuErrorCheck(hipDeviceSynchronize());
    }

    gpuErrorCheck(hipMemcpy(&tmp.data[0], data, sizeof(double) * n * n * 2, hipMemcpyDeviceToHost));
    for (uint64_t i = 0; i < n; ++i) {
        for (uint64_t j = 0; j < n; ++j) {
            ans(i, j) = tmp(j + n, i);
        }
    }

    gpuErrorCheck(hipFree(data));
    return ans;
}

const Matrix operator* (const Matrix &m1, const Matrix &m2) {
    Matrix ans(m1.n);
    for (uint64_t i = 0; i < m1.n; ++i) {
        for (uint64_t j = 0; j < m1.m; ++j) {
            double tmp = 0;
            for (uint64_t k = 0; k < m1.m; ++k) {
                tmp += m1(i, k) * m2(k, j);
            }
            ans(i, j) =  tmp;
        }
    }
    return ans;
}

std::istream &operator>> (std::istream &input, Matrix &matrix) {
    for (uint64_t i = 0; i < matrix.n; ++i) {
        for (uint64_t j = 0; j < matrix.m; ++j) {
            input >> matrix.data[i * matrix.m + j];
        }
    }
    return input;
}

std::ostream &operator<< (std::ostream &output, const Matrix &matrix) {
    for (uint64_t i = 0; i < matrix.n; ++i) {
        for (uint64_t j = 0; j < matrix.m - 1; ++j) {
            output << matrix.data[i * matrix.m + j] << ' ';
        }
        output << matrix.data[(i + 1) * matrix.m - 1] << '\n';
    }
    return output;
}